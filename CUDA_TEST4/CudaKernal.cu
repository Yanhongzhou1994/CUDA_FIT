#include "hip/hip_runtime.h"
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipblas.h"
#include "pch.h"
//#include "stdafx.h"
#include "cv.h"
//#include <process.h>
//#include "CameraApi.h"
#include "LaserRange.h"
//#include "afxwin.h"
//#include "windows.h"
#include "math.h"
//#include "cstdlib"
//#include "sstream"
//#include "ImProcess.h"
//#include "opencv2/core/core.hpp"
//#include "opencv2/imgproc/imgproc.hpp"
//#include "opencv2/calib3d/calib3d.hpp"
#include "CudaTest.h"
#include <>
#include <hip/device_functions.h>

//#include <stdio.h>
//#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
//#include "CudaKernal.cuh"

using namespace std;
using namespace cv;
using namespace cv::cuda;

//extern "C" void GetGaussFitCuda(GpuMat gpuMat, MPoint *point, double maxError, double minError, int yRange, int Colonce);

//����˷� a[M][N]*b[N][S]
hipError_t checkCudaError(hipError_t CudaFunction,const char* ident) {
	hipError_t err = CudaFunction;
	if (err != hipSuccess) {
		fprintf(stderr, "%s \t hipError_t:%s\n",ident,hipGetErrorString(hipGetLastError()));
	}
	return err;
}

//Coloncel��ɨ��õ�洢
__global__ void GetGaussPointCuda(PtrStepSz<uchar1> src, MPoint *point, int **gpu_data, int Colonce, int Rows, int Cols) {
	int threadId = threadIdx.x;
	//printf("%d\n",threadId);
	//__shared__ int *gpu_cr;
	//gpu_cr = new int [Rows*Cols];
	//���д�������
	for (int j = 0; j < Rows; j++)
	{
		for (int i = 0; i < Colonce; i++)
		{
			gpu_data[i + threadId * Colonce][j] = (int)src(j, threadId*Colonce + i).x;
		}
	}
	//int i = 0, j = 0;
	//gpu_cr[i*Rows + j] = *((int*)&src( j, threadId*Colonce + i));

	//ȡÿ�����ֵλ��
	for (int i = 0; i < Colonce; i++) {
		int MaxPixel = gpu_data[i + threadId * Colonce][0];
		//printf("the first pixel is %d \n", MaxPixel);
		int MaxY = 0;
		for (int j = 1; j < Rows; j++)
		{
			if (gpu_data[i + threadId * Colonce][j] > MaxPixel)
			{
				MaxPixel = gpu_data[i + threadId * Colonce][j];
				MaxY = j;
			}
		}

		point[threadId*Colonce + i].x = threadId * Colonce + i;
		point[threadId*Colonce + i].y = MaxY;
		point[threadId*Colonce + i].bright = MaxPixel;
	}
	__syncthreads();
}

//����ɸѡ�������˹��
__global__ void GetGaussFitRes(MPoint *point, int **gpu_data, double maxError, double minError, int yRange, int Rows, int Cols) 
{
	//ͨ���鲢�н��һ��block��thread�����õ�����
	int threadId = blockIdx.x*blockDim.x + threadIdx.x;
	//�ж���ȷ�����߳��пɴ�������
	if (threadId < Cols)
	{
		////��˹��洢����
		//int *y; //�洢��˹����ÿ�е���λ��
		//int *br; //�洢��˹���ֵ
		GPoint *gpoint = new GPoint[2 * yRange];
		int Pixnum = 0; //ͳ�Ƹ�˹�����
		//ȷ�����½�λ�� ���ټ������
		double minLine = minError * point[threadId].bright;
		double maxLine = (1-maxError) * point[threadId].bright;
		//��˹��ɸѡ
		for (int i = (point[threadId].y - yRange); i < (point[threadId].y+yRange+1); i++)
		{
			if ((gpu_data[threadId][i] > minLine)&&(gpu_data[threadId][i] < maxLine))
			{
				gpoint[Pixnum].x = i;
				gpoint[Pixnum].brightness = gpu_data[threadId][i];
				Pixnum++;
			}
		}
		point[threadId].Pixnum = Pixnum;
		/*
		if (Pixnum > 3) {
			//�����������
			int n = Pixnum;
			//X����1 x x^2��  n*3
			double **X = new double*[n];
			for (int i = 0; i < n; i++) {
				X[i] = new double[3];
			}
			//XT���� X��ת��  3*n
			double **XT = new double*[3];
			for (int i = 0; i < 3; i++) {
				XT[i] = new double[n];
			}
			//Z����(brightness) n*1
			double *Z = new double[n];
			//B���󣨽���� 3*1
			double *B = new double[3];
			//SA���� ��XT*X�� 3*3
			double **SA = new double*[3];
			for (int i = 0; i < 3; i++) {
				SA[i] = new double[3];
			}
			//SAN���� SA�������  3*3
			double **SAN = new double*[3];
			for (int i = 0; i < 3; i++) {
				SAN[i] = new double[3];
			}
			//SC����  SAN*XT 3*n
			double **SC = new double *[3];
			for (int i = 0; i < 3; i++) {
				SC[i] = new double[n];
			}
			//����X�����Z���� ˳�ִ���ת��XT
			for (int i = 0; i < n; i++) {
				X[i][0] = 1;
				X[i][1] = gpoint[i].x;
				X[i][2] = gpoint[i].x*gpoint[i].x;
				Z[i] = gpoint[i].brightness;
				XT[0][i] = 1;
				XT[1][i] = gpoint[i].x;
				XT[2][i] = gpoint[i].x*gpoint[i].x;
			}
			//����XT*X
			for (int m = 0; m < 3; m++) {
				for (int s = 0; s < 3; s++) {
					SA[m][s] = 0;
					for (int i = 0; i < n; i++) {
						SA[m][s] += XT[m][i] * X[i][s];
					}
				}
			}
			//����SAN
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					SAN[i][j] = 0;
				}
			}
			//����SC = SAN*XT
			for (int m = 0; m < 3; m++) {
				for (int s = 0; s < n; s++) {
					SC[m][s] = 0;
					for (int i = 0; i < 3; i++) {
						SC[m][s] += SAN[m][i] * XT[i][s];
					}
				}
			}
			//����B = SC*Z
			for (int m = 0; m < 3; m++) {
				B[m] = 0;
				for (int i = 0; i < n; i++) {
					B[m] += SC[m][i] * Z[i];
				}
			}
			//����B
			point[threadId].cx = threadId;
			point[threadId].cy = (-B[1]) / (2 * B[2]);
			point[threadId].bright = exp(B[0] - B[1] * B[1] / (4 * B[2]));

		}
		else
		{
			point[threadId].cx = threadId;
			point[threadId].cy = 0;
			point[threadId].bright = 0;
		}
		
		*/
		
		delete[] gpoint;

	}
	else
		return;

}
	
	//��˹��ɸѡ
	//for (int i = 0; i < Colonce; i++)
	//{
	//	int Pixnum = 0;
	//	//GPoint *gpoint;
	//	//point[threadId*Colonce+i].gpoint = new GPoint[Rows];
	//	//point[i].gpoint = new GPoint[Rows];
	//	for (int j = 0; j < Rows; j++)
	//	{
	//		if ((gpu_cr[Rows*i + j] > minError*point[threadId*Colonce + i].bright)
	//			&& (gpu_cr[Rows*i + j] < (1 - maxError)*point[threadId*Colonce + i].bright)
	//			&& (abs(j - point[threadId*Colonce + i].y) < yRange))
	//		{
	//			point[threadId*Colonce + i].gpoint[Pixnum].x = threadId * Colonce + i;
	//			point[threadId*Colonce + i].gpoint[Pixnum].brightness = gpu_cr[Rows*i + j];
	//			Pixnum++;
	//		}
	//		if ((j - point[threadId*Colonce + i].y) < yRange)
	//			break;
	//	}
	//	point[threadId*Colonce + i].Pixnum = Pixnum;

		/*
		//��������
		if (Pixnum >= 3)
		{
			__shared__ int *X;
			X = new int[Pixnum * 3];
			__shared__ int *Z;
			Z = new int[Pixnum];
			//��������<3.5 ����Ƕ�ײ��к˺���
			//dim3 blockSEX(1, 0, 0);
			//dim3 threadSEX(Pixnum, 0, 0);
			//����X��Z����
			//SetElementX << <blockSEX, threadSEX >> > (gpoint, X, Pixnum);
			//����X����(n*3) Z����n*1)
			for (int i = 0; i < Pixnum; i++)
			{
				for (int j = 0; j < 3; j++)
				{
					if (j = 0)
					{
						X[i * 3 + j] = 1;
					}
					if (j = 1)
					{
						X[i * 3 + j] = gpoint[i].x;
					}
					if (j = 2)
					{
						X[i * 3 + j] = gpoint[i].x*gpoint[i].x;
					}
				}
				Z[i] = gpoint[i].brightness;
			}
			//��Xת��
			__shared__ int *XT;
			XT = new int[Pixnum* 3];
			for (int i = 0; i < 3; i++)
			{
				for (int j = 0; j < Pixnum; j++)
				{
					XT[i*Pixnum + j] = X[j * 3 + i];
				}
			}
			//��XT*X���
			__shared__ int *SA;
			SA = new int[3 * 3];
			for (int m = 0; i < 3; i++)
			{
				for (int s = 0; s < 3; s++)
				{
					for (int n = 0; n < Pixnum; n++)
					{
						SA[m * 3 + s] = XT[m*Pixnum + n] * X[n * 3 + s];
					}
				}
			}
			//��SA�����
			__shared__ int *SAN;
			SAN = new int[3 * 3];


		}*/
	//}
	//delete &gpu_cr;
	

//#define N 3
//__global__ void MatAdd(const int **A, const int **B, int **C)
//{
//	int i = threadIdx.x;
//	int j = threadIdx.y;
//	C[i][j] = A[i][j] + B[i][j];
//	//__syncthreads();
//}



//extern "C" void GetGaussFitCuda(GpuMat gpuMat, MPoint *point, double maxError, double minError, int yRange, int Colonce);

extern "C"
void CudaGuassHC(Mat matImage, MPoint *point, double maxError, double minError, int yRange, int Colonce) {

	int Rows = matImage.rows;
	int Cols = matImage.cols;// *matImage.channels();
	//InputArray inputMat(matImage);
	//for (int j = 0; j < Rows; j++) {
	//	//uchar* data = gpuMat.ptr<uchar>(j);
	//	for (int i = 0; i < Cols; i++) {
	//		int datt = inputMat.ptr<uchar>(j)[i];
	//		//cout << "(" << i << "," <<j << "):" << datt << endl;
	//		printf("(%d,%d):%d\n", i, j, datt);
	//	}
	//}
	//cout << Cols << endl;
	GpuMat gpuMat(matImage);
	//gpuMat.upload(matImage);
	//for (int j = 0; j < Rows; j++) {
	//	//uchar* data = gpuMat.ptr<uchar>(j);
	//	for (int i = 0; i < Cols; i++) {
	//		int datt = gpuMat.ptr<uchar>(j)[i];
	//		//cout << "(" << i << "," <<j << "):" << datt << endl;
	//		printf("(%d,%d):%d\n", i, j, datt);
	//	}
	//}
	//�ṹ��ָ���ϴ�
	MPoint *gpu_point;
	//gpu_point = new MPoint[Cols];	
	checkCudaError(hipMalloc((void**)&gpu_point, sizeof(MPoint)*Cols), "malloc error1");
	//�Դ�ͼ�񻺴����
	int **gpu_data;
	int *gpu_data_d;
	int **cpu_data = (int**)malloc(sizeof(int*)*Cols);
	int *cpu_data_d = (int*)malloc(sizeof(int)*Cols*Rows);
	checkCudaError(hipMalloc((void**)&gpu_data, Cols * sizeof(int**)), "malloc error2");
	checkCudaError(hipMalloc((void**)&gpu_data_d, Cols *Rows * sizeof(int)), " malloc error2");
	for (int i = 0; i < Cols; i++) {
		cpu_data[i] = gpu_data_d + Rows * i;
		//�׵�ַ��ֵ ��һά����תΪ��ά
	}
	checkCudaError(hipMemcpy(gpu_data, cpu_data, sizeof(int*)*Cols, hipMemcpyHostToDevice), "memcpy error1");
	checkCudaError(hipMemcpy(gpu_data_d, cpu_data_d, sizeof(int)*Rows*Cols, hipMemcpyHostToDevice), "memcpy error1");  

	/*if (hipSuccess != hipMemcpy(gpu_point, point, sizeof(point)*Cols, hipMemcpyHostToDevice)) {
		printf("cuda memcpy up error1!\n");
	}*/
	
	//dim3 threads_all(Cols / Colonce);
	//ÿcolonce��ͳһ���� 
	GetGaussPointCuda << <1, Cols/Colonce >> > (gpuMat, gpu_point, gpu_data, Colonce, Rows, Cols);
	hipDeviceSynchronize();
	//�滮������  ֮�����Ϊֻ�滮һ��
	int Blocknum, Threadnum;
	if (Cols > 1024) {
		Blocknum = Cols / 1024 + 1;
		Threadnum = 1024;
	}
	else {
		Blocknum = 1;
		Threadnum = Cols;
	}
	//���и�˹���
	GetGaussFitRes << <Blocknum, Threadnum >> > (gpu_point, gpu_data, maxError, minError, yRange, Rows, Cols);
	checkCudaError(hipMemcpy(point, gpu_point, sizeof(MPoint)*Cols, hipMemcpyDeviceToHost), "memcpy down error1");
	for (int i = 0; i < Cols; i++)
	{
		//cout << "("<<point[i].x<<","<< point[i].y<<"):"<< point[i].bright << endl;
		printf("(%d,%d):%d\t, here are %d GaussPoints\n", point[i].x, point[i].y, point[i].bright,point[i].Pixnum);
	}

	/*for (int i = 0; i < Cols; i++)
	{
		free((void*)cpu_data[i]);
	}*/
	free((void*)cpu_data);
	free(cpu_data_d);
	/*for (int i = 0; i < Cols; i++)
	{
		hipFree((void*)gpu_data[i]);
	}*/
	hipFree(gpu_data);
	hipFree(gpu_point);
	hipFree(gpu_data_d);
	gpuMat.release();



}

extern "C" void GuassFitGpuHcT(Mat matImage, MPoint *point, double maxError, double minError, int yRange, int Colonce)
{

}

